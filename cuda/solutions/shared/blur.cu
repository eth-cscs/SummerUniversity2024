#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "util.hpp"
#include "hip_stream.hpp"
#include "cuda_event.hpp"

template <int Threads>
__global__
void blur_shared_block(const double *in, double* out, int n) {
    __shared__ double buffer[Threads+2];

    auto block_start = blockDim.x * blockIdx.x;
    auto li = threadIdx.x + 1;
    auto gi = li + block_start;

    if(gi<n-1) {
        // load shared memory
        buffer[li] = in[gi];
        if(li==1) {
            buffer[0] = in[block_start];
            buffer[Threads+1] = in[block_start+Threads+1];
        }

        __syncthreads();

        out[gi] = 0.25*(buffer[li-1] + 2.0*buffer[li] + buffer[li+1]);
    }
}

__global__
void blur_shared(const double *in, double* out, int n) {
    extern __shared__ double buffer[];

    auto block_start = blockDim.x * blockIdx.x;
    auto li = threadIdx.x + 1;
    auto gi = li + block_start;

    if(gi<n-1) {
        // load shared memory
        buffer[li] = in[gi];
        if(li==1) {
            buffer[0] = in[block_start];
            buffer[blockDim.x+1] = in[block_start+blockDim.x+1];
        }

        __syncthreads();

        out[gi] = 0.25*(buffer[li-1] + 2.0*buffer[li] + buffer[li+1]);
    }
}

// No shared memory used, but 3 reads and 1 write to global memory
__global__
void blur(const double *in, double* out, int n) {
    auto i = threadIdx.x + blockDim.x * blockIdx.x + 1;

    if(i<n-1) {
        out[i] = 0.25*(in[i-1] + 2.0*in[i] + in[i+1]);
    }
}

// No shared memory used, but 1 read and 3 writes to global memory
__global__
void blur_alt(const double *in, double* out, int n) {
    auto i = threadIdx.x + blockDim.x * blockIdx.x;
    auto in_i = in[i];

    if(i > 0 && i < n-1) 
    {
        out[i-1] += 0.25*in_i;
        out[i] += 0.5*in_i; 
        out[i+1] += 0.25*in_i;
    }
    if (i == 0)
    {
        out[i+1] += 0.25*in_i;
    }
    if (i == n-1)
    {
        out[i-1] += 0.25*in_i;
    }
}

int main(int argc, char** argv) {
    size_t pow    = read_arg(argc, argv, 1, 20);
    size_t nsteps = read_arg(argc, argv, 2, 100);
    bool use_shared = read_arg(argc, argv, 3, false);
    size_t n = 1 << pow;

    const auto size_in_bytes = n * sizeof(double);

    std::cout << "-- blur 1D test of length n = " << n << " : " << size_in_bytes*1e-9 << "MB\n";
    std::cout << "-- using " << (use_shared ? "shared": "direct") << " kernel\n";

    auto x_host = malloc_host<double>(n+2, 0.);
    // set boundary conditions to 1
    x_host[0]   = 1.0;
    x_host[n+1] = 1.0;

    auto x0 = malloc_device<double>(n+2);
    auto x1 = malloc_device<double>(n+2);

    // copy initial conditions to device
    copy_to_device<double>(x_host, x0, n+2);
    copy_to_device<double>(x_host, x1, n+2);

    // find the launch grid configuration
    constexpr auto block_dim = 128;
    const auto grid_dim = (n+(block_dim-1))/block_dim;

    // perform nsteps of the blur operation
    hip_stream stream;
    auto start_event = stream.enqueue_event();
    for(auto step=0; step<nsteps; ++step) {
        if (use_shared) {
            blur_shared_block<block_dim><<<grid_dim, block_dim>>>(x0, x1, n);
            // blur_shared<<<grid_dim, block_dim, (block_dim+2)*sizeof(double)>>>(x0, x1, n);
        }
        else {
            blur<<<grid_dim, block_dim>>>(x0, x1, n);
            // blur_alt<<<grid_dim, block_dim>>>(x0, x1, n);
        }
        std::swap(x0, x1);
    }
    auto stop_event = stream.enqueue_event();

    // copy result back to host
    copy_to_host<double>(x0, x_host, n+2);

    stop_event.wait();
    auto time = stop_event.time_since(start_event);
    std::cout << "==== " << time << " seconds : " << 1e3*time/nsteps << " ms/step\n";

    return 0;
}

