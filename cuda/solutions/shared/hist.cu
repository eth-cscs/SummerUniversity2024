#include "hip/hip_runtime.h"
#include <numeric>

#include <cstdio>
#include <hip/hip_runtime.h>

#include "util.hpp"

__global__
void histogram(int* x, int* bins, int n) {
    auto i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i<n) {
        const auto c = x[i];
        //bins[c]++;
        atomicAdd(bins+c, 1);
    }
}

int main(void) {
    const int n = 1024;
    const int c = 16;

    int* x = malloc_managed<int>(n);
    for (auto i=0; i<n; ++i) x[i] = rand()%c;

    int* bins = malloc_managed<int>(c);
    std::fill(bins, bins+c, 0);

    histogram<<<1, n>>>(x, bins, n);
    hipDeviceSynchronize();

    printf("bins: ");
    for (auto i=0; i<c; ++i) printf("%d ", bins[i]); printf("\n");

    auto sum = std::accumulate(bins, bins+c, 0);
    printf("sum %d, expected %d\n", sum, n);

    hipFree(x);
    hipFree(bins);
    return 0;
}
