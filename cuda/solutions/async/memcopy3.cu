#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "util.hpp"
#include "hip_stream.hpp"
#include "cuda_event.hpp"

#define USE_PINNED
// CUDA kernel implementing newton solve for
//      f(x) = 0
// where
//      f(x) = exp(cos(x)) - 2
__global__
void newton(int n, double *x) {
    auto tid = threadIdx.x + blockDim.x * blockIdx.x;

    auto f  = [] (double x) {
        return exp(cos(x))-2;
    };
    auto fp = [] (double x) {
        return -sin(x) * exp(cos(x));
    };

    if(tid<n) {
        auto x0 = x[tid];
        for(int i=0; i<10; ++i) {
            x0 -= f(x0)/fp(x0);
        }
        x[tid] = x0;
    }
}

int main(int argc, char** argv) {
    size_t pow        = read_arg(argc, argv, 1, 20);
    size_t num_chunks = read_arg(argc, argv, 2, 1);

    size_t N = 1 << pow;
    auto size_in_bytes = N * sizeof(double);

    std::cout << "memory copy overlap test of length N = " << N
              << " : " << size_in_bytes/(1024.*1024.) << "MB"
              << " with " << num_chunks << " chunks"
              << std::endl;

    hipInit(0);

    double* xd = malloc_device<double>(N);
    double* xh = malloc_pinned<double>(N, 1.5);
    double* x  = malloc_pinned<double>(N);

    int chunk_size = N/num_chunks; // assume N % num_chunks == 0

    // precompute kernel launch configuration
    auto block_dim = 128;
    auto grid_dim = (chunk_size-1)/block_dim + 1;

    hip_stream D2H_stream;
    hip_stream H2D_stream;
    hip_stream kernel_stream;

    auto start_event = D2H_stream.enqueue_event();
    for(int i=0; i<num_chunks; ++i) {
        auto offset = i*chunk_size;

        // copy chunk to device
        copy_to_device_async<double>(xh+offset, xd+offset,
                                     chunk_size, H2D_stream.stream());

        // force the kernel stream to wait for the memcpy
        auto H2D_event = H2D_stream.enqueue_event();
        kernel_stream.wait_on_event(H2D_event);

        // solve N nonlinear problems, i.e. find x[i] s.t. f(x[i])=0
        newton<<<grid_dim, block_dim, 0, kernel_stream.stream()>>>
            (chunk_size, xd+offset);
        cuda_check_last_kernel("newton kernel");

        // copy chunk of result back to host
        auto kernel_event = kernel_stream.enqueue_event();
        D2H_stream.wait_on_event(kernel_event);
        copy_to_host_async<double>(xd+offset, x+offset,
                                   chunk_size, D2H_stream.stream());
    }
    auto end_event = D2H_stream.enqueue_event();
    end_event.wait();

    auto time_total = end_event.time_since(start_event);

    std::cout << "-------\ntimings\n-------" << std::endl;
    std::cout << "total : " << time_total << std::endl;

    // check for errors
    auto f  = [] (double x) { return exp(cos(x))-2.; };
    auto errors = 0;
    for(auto i=0; i<N; ++i) {
        if(std::fabs(f(x[i]))>1e-10) {
            errors++;
        }
    }
    if(errors>0) std::cout << "\n============ FAILED with " << errors << " errors" << std::endl;
    else         std::cout << "\n============ PASSED" << std::endl;

    hipFree(xd);
    hipHostFree(xh);
    hipHostFree(x);

    return 0;
}

