#include "hip/hip_runtime.h"
// linear algebra subroutines
// Ben Cumming @ CSCS

#include <iostream>

#include <cmath>
#include <cstdio>

#include "linalg.h"
#include "operators.h"
#include "stats.h"
#include "data.h"

namespace linalg {

namespace kernels {

// TODO implement the missing linalg kernels
__global__
void add_scaled_diff(
        double *y,
        const double* x,
        const double alpha,
        const double *l,
        const double *r,
        const int n)
{
    auto i = threadIdx.x + blockDim.x*blockIdx.x;
    if(i < n) {
        y[i] = x[i] + alpha * (l[i] - r[i]);
    }
}

__global__
void copy(double *y, const double* x, int n) {
    auto i = threadIdx.x + blockDim.x*blockIdx.x;
    if(i < n) {
        y[i] = x[i];
    }
}

// sets x := value
__global__
void fill(double* x, const double value, int n) {
    auto i = threadIdx.x + blockDim.x*blockIdx.x;
    if(i < n) {
        x[i] = value;
    }
}

// computes y := alpha*x + y
__global__
void axpy(double* y, const double alpha, const double* x, int n){
    auto i = threadIdx.x + blockDim.x*blockIdx.x;
    if(i < n) {
        y[i] += alpha*x[i];
    }
}

// computes y = alpha*(l-r)
__global__
void scaled_diff(double* y, const double alpha, const double* l, const double* r, int n) {
    auto i = threadIdx.x + blockDim.x*blockIdx.x;
    if(i < n) {
        y[i] = alpha*(l[i]-r[i]);
    }
}

// computes y := alpha*x
__global__
void scale(double* y, const double alpha, double* x, int n) {
    auto i = threadIdx.x + blockDim.x*blockIdx.x;
    if(i < n) {
        y[i] = alpha*x[i];
    }
}

// computes linear combination of two vectors y := alpha*x + beta*z
__global__
void lcomb(double* y, const double alpha, double* x, const double beta, const double* z, int n) {
    auto i = threadIdx.x + blockDim.x*blockIdx.x;
    if(i < n) {
        y[i] = alpha*x[i] + beta*z[i];
    }
}

} // namespace kernels

bool cg_initialized = false;
Field r;
Field Ap;
Field p;
Field Fx;
Field Fxold;
Field v;
Field xold;

// block dimensions for blas 1 calls
const int block_dim = 192;

int calculate_grid_dim(const int block_dim, int n) {
    return (n-1)/block_dim + 1;
}

using namespace operators;
using namespace stats;
using data::Field;

// initialize temporary storage fields used by the cg solver
// I do this here so that the fields are persistent between calls
// to the CG solver. This is useful if we want to avoid malloc/free calls
// on the device for the OpenACC implementation
void cg_init(int nx, int ny)
{
    Ap.init(nx,ny);
    r.init(nx,ny);
    p.init(nx,ny);
    Fx.init(nx,ny);
    Fxold.init(nx,ny);
    v.init(nx,ny);
    xold.init(nx,ny);

    cg_initialized = true;
}

////////////////////////////////////////////////////////////////////////////////
//  blas level 1 reductions
////////////////////////////////////////////////////////////////////////////////

// TODO implement the dot product with cublas
// HINT : use cublas_handle() to get the cublas handle

// computes the inner product of x and y
// x and y are vectors
double ss_dot(Field const& x, Field const& y)
{
    double result = 0.;
    const int n = x.length();
    hipblasDdot (cublas_handle(), n,
                x.device_data(), 1,
                y.device_data(), 1,
                &result);

    return result;
}

// computes the 2-norm of x
// x is a vector
double ss_norm2(Field const& x) {
    double result = 0;
    const int n = x.length();

    hipblasDnrm2 (cublas_handle(), n,
                x.device_data(), 1,
                &result);
    return result;
}

////////////////////////////////////////////////////////////////////////////////
//  blas level 1 vector-vector operations
////////////////////////////////////////////////////////////////////////////////

// computes y = x + alpha*(l-r)
// y, x, l and r are vectors
// alpha is a scalar
void ss_add_scaled_diff(Field& y, Field const& x, const double alpha,
    Field const& l, Field const& r)
{
    const int n = y.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::add_scaled_diff<<<grid_dim, block_dim>>>
        (y.device_data(), x.device_data(), alpha, l.device_data(), r.device_data(), n);
}

// copy one vector into another y := x
// x and y are vectors of length N
void ss_copy(Field& y, Field const& x)
{
    const int n = x.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::copy<<<grid_dim, block_dim>>>
        (y.device_data(), x.device_data(), n);
}

// TODO : implement the wrappers for
// ss_fill
// ss_axpy
// ss_scaled_diff
// ss_scale
// ss_lcomb

// sets x := value
// x is a vector
// value is a scalar
void ss_fill(Field& x, const double value)
{
    const int n = x.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::fill<<<grid_dim, block_dim>>>
        (x.device_data(), value, n);
}

// computes y := alpha*x + y
// x and y are vectors
// alpha is a scalar
void ss_axpy(Field& y, const double alpha, Field const& x)
{
    const int n = y.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::axpy<<<grid_dim, block_dim>>>
        (y.device_data(), alpha, x.device_data(), n);
}

// computes y = alpha*(l-r)
// y, l and r are vectors of length N
// alpha is a scalar
void ss_scaled_diff(Field& y, const double alpha, Field const& l, Field const& r)
{
    const int n = y.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::scaled_diff<<<grid_dim, block_dim>>>
        (y.device_data(), alpha, l.device_data(), r.device_data(), n);
}

// computes y := alpha*x
// alpha is scalar
// y and x are vectors
void ss_scale(Field& y, const double alpha, Field& x)
{
    const int n = y.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::scale<<<grid_dim, block_dim>>>
        (y.device_data(), alpha, x.device_data(), n);
}

// computes linear combination of two vectors y := alpha*x + beta*z
// alpha and beta are scalar
// y, x and z are vectors
void ss_lcomb(Field& y, const double alpha, Field& x, const double beta, Field const& z)
{
    const int n = y.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::lcomb<<<grid_dim, block_dim>>>
        (y.device_data(), alpha, x.device_data(), beta, z.device_data(), n);
}

// conjugate gradient solver
// solve the linear system A*x = b for x
// the matrix A is implicit in the objective function for the diffusion equation
// the value in x constitute the "first guess" at the solution
// x(N)
// ON ENTRY contains the initial guess for the solution
// ON EXIT  contains the solution
void ss_cg(Field& x, Field const& b, const int maxiters, const double tol, bool& success)
{
    // this is the dimension of the linear system that we are to solve
    int nx = data::options.nx;
    int ny = data::options.ny;

    if(!cg_initialized) {
        cg_init(nx,ny);
    }

    // epsilon value use for matrix-vector approximation
    double eps     = 1.e-8;
    double eps_inv = 1. / eps;

    // initialize memory for temporary storage
    ss_fill(Fx,    0.0);
    ss_fill(Fxold, 0.0);
    ss_copy(xold, x);

    // matrix vector multiplication is approximated with
    // A*v = 1/epsilon * ( F(x+epsilon*v) - F(x) )
    //     = 1/epsilon * ( F(x+epsilon*v) - Fxold )
    // we compute Fxold at startup
    // we have to keep x so that we can compute the F(x+exps*v)
    diffusion(x, Fxold);

    // v = x + epsilon*x
    ss_scale(v, 1.0 + eps, x);

    // Fx = F(v)
    diffusion(v, Fx);

    // r = b - A*x
    // where A*x = (Fx-Fxold)/eps
    ss_add_scaled_diff(r, b, -eps_inv, Fx, Fxold);

    // p = r
    ss_copy(p, r);

    // rold = <r,r>
    double rold = ss_dot(r, r);
    double rnew = rold;

    // check for convergence
    success = sqrt(rold) < tol;
    if (success) {
        return;
    }

    int iter;
    for(iter=0; iter<maxiters; iter++) {
        // Ap = A*p
        ss_lcomb(v, 1.0, xold, eps, p);
        diffusion(v, Fx);
        ss_scaled_diff(Ap, eps_inv, Fx, Fxold);

        // alpha = rold / p'*Ap
        double alpha = rold / ss_dot(p, Ap);

        // x += alpha*p
        ss_axpy(x, alpha, p);

        // r -= alpha*Ap
        ss_axpy(r, -alpha, Ap);

        // find new norm
        rnew = ss_dot(r, r);

        // test for convergence
        if (sqrt(rnew) < tol) {
            success = true;
            break;
        }

        // p = r + (rnew/rold) * p
        ss_lcomb(p, 1.0, r, rnew / rold, p);

        rold = rnew;
    }
    stats::iters_cg += iter + 1;

    if (!success) {
        std::cerr << "ERROR: CG failed to converge after " << iter
                  << " iterations, with residual " << sqrt(rnew)
                  << std::endl;
    }
}

} // namespace linalg
