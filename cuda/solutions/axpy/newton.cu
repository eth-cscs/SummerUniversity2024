#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "util.hpp"

__host__ __device__
double f(double x) {
    return exp(cos(x))-2;
};

__host__ __device__
double fp(double x) {
    return -sin(x) * exp(cos(x));
};

// implements newton solve for
//      f(x) = 0
// where
//      f(x) = exp(cos(x)) - 2
void newton_host(int n, double *x) {
    for(int i=0; i<n; ++i) {
        auto x0 = x[i];
        for(int iter=0; iter<5; ++iter) {
            x0 -= f(x0)/fp(x0);
        }
        x[i] = x0;
    }
}

// TODO : implement newton_device() kernel that performs the work in newton_host
//        in parallel on the GPU
__global__
void newton_device(int n, double* x)
{
    auto i = threadIdx.x + blockDim.x*blockIdx.x;
    auto x0i = x[i];
    for (int iter = 0; iter < 5; ++iter)
    {
        x0i -= f(x0i)/fp(x0i);
    }
    x[i] = x0i;
}

int main(int argc, char** argv) {
    size_t pow        = read_arg(argc, argv, 1, 20);

    size_t n = 1 << pow;
    auto size_in_bytes = n * sizeof(double);

    std::cout << "memory copy overlap test of length n = " << n
              << " : " << size_in_bytes/1e9 << "MB\n";

    hipInit(0);

    double* xd = malloc_device<double>(n);
    double* xh = malloc_host<double>(n, 1.5);
    double* x  = malloc_host<double>(n);

    // compute kernel launch configuration
    auto block_dim = 128;
    auto grid_dim = (n+block_dim-1)/block_dim;

    auto time_h2d = -get_time();
    copy_to_device(xh, xd, n);
    time_h2d += get_time();

    hipDeviceSynchronize();
    auto time_kernel = -get_time();

    // TODO: launch kernel (use block_dim and grid_dim calculated above)
    newton_device<<<grid_dim, block_dim>>>(n, xd);

    hipDeviceSynchronize();
    time_kernel += get_time();

    auto time_d2h = -get_time();
    copy_to_host(xd, x, n);
    time_d2h += get_time();

    std::cout << "-------\ntimings\n-------\n";
    std::cout << "H2D    : " << time_h2d    << " s\n";
    std::cout << "D2H    : " << time_d2h    << " s\n";
    std::cout << "kernel : " << time_kernel << " s\n";

    // check for errors
    auto errors = 0;
    for(auto i=0; i<n; ++i) {
        if(std::fabs(f(x[i]))>1e-10) {
            errors++;
        }
    }
    if(errors>0) std::cout << "\n============ FAILED with " << errors << " errors\n";
    else         std::cout << "\n============ PASSED\n";

    hipFree(xd);
    free(xh);
    free(x);

    return 0;
}

