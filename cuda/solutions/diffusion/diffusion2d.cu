#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdio>

#include <hip/hip_runtime.h>

#include "util.hpp"
#include "hip_stream.hpp"

// 2D diffusion example
// the grid has a fixed width of nx=128
// the use specifies the height, ny, as a power of two
// note that nx and ny have 2 added to them to account for halos

template <typename T>
void fill_gpu(T *v, T value, int n);

void write_to_file(int nx, int ny, double* data);

__global__
void diffusion(double *x0, double *x1, int nx, int ny, double dt) {
    int i = threadIdx.x + blockDim.x*blockIdx.x + 1;
    int j = threadIdx.y + blockDim.y*blockIdx.y + 1;

    if (i<nx-1 && j<ny-1) {
        int pos = nx*j + i;
          x1[pos] = x0[pos] + dt * (-4.*x0[pos]
                     + x0[pos-1] + x0[pos+1]
                     + x0[pos-nx] + x0[pos+nx]);

    }
}
// TODO : implement stencil using 2d launch configuration
// NOTE : i-major ordering, i.e. x[i,j] is indexed at location [i+j*nx]
//  for(i=1; i<nx-1; ++i) {
//    for(j=1; j<ny-1; ++j) {
//        x1[i,j] = x0[i,j] + dt * (-4.*x0[i,j]
//                   + x0[i,j-1] + x0[i,j+1]
//                   + x0[i-1,j] + x0[i+1,j]);
//    }
//  }

int main(int argc, char** argv) {
    // set up parameters
    // first argument is the y dimension = 2^arg
    size_t pow    = read_arg(argc, argv, 1, 8);
    // second argument is the number of time steps
    size_t nsteps = read_arg(argc, argv, 2, 100);

    // set domain size
    size_t nx = 128+2;
    size_t ny = (1 << pow)+2;
    double dt = 0.1;

    std::cout << "\n## " << nx << "x" << ny
              << " for " << nsteps << " time steps"
              << " (" << nx*ny << " grid points)"
              << std::endl;

    // allocate memory on device and host
    // note : allocate enough memory for the halo around the boundary
    auto buffer_size = nx*ny;
    double *x_host = malloc_host<double>(buffer_size);
    double *x0     = malloc_device<double>(buffer_size);
    double *x1     = malloc_device<double>(buffer_size);

    // set initial conditions of 0 everywhere
    fill_gpu(x0, 0., buffer_size);
    fill_gpu(x1, 0., buffer_size);

    // set boundary conditions of 1 on south border
    fill_gpu(x0, 1., nx);
    fill_gpu(x1, 1., nx);
    fill_gpu(x0+nx*(ny-1), 1., nx);
    fill_gpu(x1+nx*(ny-1), 1., nx);

    hip_stream stream;
    hip_stream copy_stream();
    auto start_event = stream.enqueue_event();

    // grid and block config
    auto find_num_blocks = [](int x, int bdim) {return (x+bdim-1)/bdim;};
    dim3 block_dim(16, 16);
    int nbx = find_num_blocks(nx-2, block_dim.x);
    int nby = find_num_blocks(ny-2, block_dim.y);
    dim3 grid_dim(nbx, nby);

    // time stepping loop
    for(auto step=0; step<nsteps; ++step) {
        // TODO: launch the diffusion kernel in 2D
        diffusion<<<grid_dim, block_dim>>>(x0, x1, nx, ny, dt);
        std::swap(x0, x1);
    }
    auto stop_event = stream.enqueue_event();
    stop_event.wait();

    copy_to_host<double>(x0, x_host, buffer_size);

    double time = stop_event.time_since(start_event);

    std::cout << "## " << time << "s, "
              << nsteps*(nx-2)*(ny-2) / time << " points/second"
              << std::endl << std::endl;

    std::cout << "writing to output.bin/bov" << std::endl;
    write_to_file(nx, ny, x_host);

    return 0;
}

template <typename T>
__global__
void fill(T *v, T value, int n) {
    int tid  = threadIdx.x + blockDim.x*blockIdx.x;

    if(tid<n) {
        v[tid] = value;
    }
}

template <typename T>
void fill_gpu(T *v, T value, int n) {
    auto block_dim = 192ul;
    auto grid_dim = n/block_dim + (n%block_dim ? 1 : 0);

    fill<T><<<grid_dim, block_dim>>>(v, value, n);
}

void write_to_file(int nx, int ny, double* data) {
    {
        FILE* output = fopen("output.bin", "w");
        fwrite(data, sizeof(double), nx * ny, output);
        fclose(output);
    }

    std::ofstream fid("output.bov");
    fid << "TIME: 0.0" << std::endl;
    fid << "DATA_FILE: output.bin" << std::endl;
    fid << "DATA_SIZE: " << nx << " " << ny << " 1" << std::endl;;
    fid << "DATA_FORMAT: DOUBLE" << std::endl;
    fid << "VARIABLE: phi" << std::endl;
    fid << "DATA_ENDIAN: LITTLE" << std::endl;
    fid << "CENTERING: nodal" << std::endl;
    fid << "BRICK_SIZE: 1.0 1.0 1.0" << std::endl;
}
