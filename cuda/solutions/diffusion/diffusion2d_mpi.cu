#include "hip/hip_runtime.h"
#include <cstdlib>

#include <fstream>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>

#include <mpi.h>

#include "util.hpp"
#include "hip_stream.hpp"

// 2D diffusion example with mpi
// the grid has a fixed width of nx=128
// the use specifies the height, ny, as a power of two
// note that nx and ny have 2 added to them to account for halos
//
// the domain decomposition is in the vertical
// ny is the height of the local sub-domain

void write_to_file(int nx, int ny, double* data, int mpi_size, int mpi_rank);

template <typename T>
void fill_gpu(T *v, T value, int n);

__global__
void diffusion(double *x0, double *x1, int nx, int ny, double dt) {
    auto i = threadIdx.x + blockIdx.x*blockDim.x+1;
    auto j = threadIdx.y + blockIdx.y*blockDim.y+1;

    if (i<nx-1 && j<ny-1) {
        auto pos = i + j*nx;
        x1[pos] = x0[pos] + dt * (-4.*x0[pos]
                   + x0[pos-nx] + x0[pos+nx]
                   + x0[pos-1]  + x0[pos+1]);
    }
}

int main(int argc, char** argv) {
    // set up parameters
    // first argument is the y dimension = 2^arg
    size_t pow    = read_arg(argc, argv, 1, 8);
    // second argument is the number of time steps
    size_t nsteps = read_arg(argc, argv, 2, 100);

    // set domain size
    size_t nx = 128;
    size_t ny = 1 << pow;
    double dt = 0.1;

    // initialize MPI
    int mpi_rank, mpi_size;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);

    bool use_rdma = (nullptr!=std::getenv("MPICH_RDMA_ENABLED_CUDA"));

    // calculate global domain sizes
    if(ny%mpi_size) {
        std::cout << "error : global domain dimension " << ny
                  << "must be divisible by number of MPI ranks " << mpi_size
                  << std::endl;
        exit(1);
    }
    else if(mpi_rank==0) {
        std::cout << "------------------------------------------\n";
        std::cout << mpi_size << " MPI ranks, "
                  << (use_rdma? "with RDMA\n": "no RDMA\n");
        std::cout << nx << "x" << ny
                  << " : " << nx << "x" << ny/mpi_size << " per rank"
                  << " for " << nsteps << " time steps"
                  << " (" << nx*ny << " grid points)"
                  << std::endl;
    }
    ny /= mpi_size;

    // adjust dimensions for halo
    nx += 2;
    ny += 2;

    // allocate memory on device and host
    // note : allocate enough memory for the halo around the boundary
    auto buffer_size = nx*ny;
    double *x_host = malloc_pinned<double>(buffer_size);
    double *x0     = malloc_device<double>(buffer_size);
    double *x1     = malloc_device<double>(buffer_size);

    // set initial conditions of 0 everywhere
    fill_gpu(x0, 0., buffer_size);
    fill_gpu(x1, 0., buffer_size);

    // set boundary conditions of 1 on south border
    if(mpi_rank==0) { // south boundary
        fill_gpu(x0, 1., nx);
        fill_gpu(x1, 1., nx);
    }
    if(mpi_rank==mpi_size-1) { // north boundary
        fill_gpu(x0+nx*(ny-1), 1., nx);
        fill_gpu(x1+nx*(ny-1), 1., nx);
    }

    hip_stream stream;
    hip_stream copy_stream();
    auto start_event = stream.enqueue_event();

    const dim3 block_dim(16, 16);
    const dim3 grid_dim(
            (nx-3)/block_dim.x+1,
            (ny-3)/block_dim.y+1);

    MPI_Status status_north;
    MPI_Status status_south;

    auto recv_buffer = malloc_pinned<double>(nx);
    auto send_buffer = malloc_pinned<double>(nx);

    // time stepping loop
    for(auto step=0; step<nsteps; ++step) {

        // perform halo exchange
        // x0(:, 0)    <- south
        // x0(:, 1)    -> south
        // x0(:, ny-1) <- north
        // x0(:, ny-2) -> north
        if (use_rdma) {
            if (mpi_rank>0) {
                 MPI_Sendrecv(x0+nx, nx, MPI_DOUBLE,
                            mpi_rank-1, 0,
                            x0, nx, MPI_DOUBLE,
                            mpi_rank-1, 1,
                            MPI_COMM_WORLD, &status_south);
            }
            if (mpi_rank<mpi_size-1) {
                 MPI_Sendrecv(x0+(ny-2)*nx, nx, MPI_DOUBLE,
                            mpi_rank+1, 1,
                            x0+(ny-1)*nx, nx, MPI_DOUBLE,
                            mpi_rank+1, 0,
                            MPI_COMM_WORLD, &status_north);
            }
        }
        else {
            if (mpi_rank>0) {
                 copy_to_host(x0+nx, send_buffer, nx);
                 MPI_Sendrecv(send_buffer, nx, MPI_DOUBLE,
                            mpi_rank-1, 0,
                            recv_buffer, nx, MPI_DOUBLE,
                            mpi_rank-1, 1,
                            MPI_COMM_WORLD, &status_south);
                 copy_to_device(recv_buffer, x0, nx);
            }
            if (mpi_rank<mpi_size-1) {
                 copy_to_host(x0+(ny-2)*nx, send_buffer, nx);
                 MPI_Sendrecv(send_buffer, nx, MPI_DOUBLE,
                            mpi_rank+1, 1,
                            recv_buffer, nx, MPI_DOUBLE,
                            mpi_rank+1, 0,
                            MPI_COMM_WORLD, &status_north);
                 copy_to_device(recv_buffer, x0+(ny-1)*nx, nx);
            }
        }
        diffusion<<<grid_dim, block_dim>>>(x0, x1, nx, ny, dt);

        std::swap(x0, x1);
    }
    auto stop_event = stream.enqueue_event();
    stop_event.wait();

    copy_to_host<double>(x0, x_host, buffer_size);

    double time = stop_event.time_since(start_event);

    if(mpi_rank==0) {
        std::cout << "time " << time << " s, "
                  << nsteps*(nx-2)*(ny-2)*mpi_size / time << " points/second"
                  << std::endl;
    }
    write_to_file(nx, ny, x_host, mpi_size, mpi_rank);

    MPI_Finalize();

    return 0;
}

template <typename T>
__global__
void fill(T *v, T value, int n) {
    int tid  = threadIdx.x + blockDim.x*blockIdx.x;

    if(tid<n) {
        v[tid] = value;
    }
}

template <typename T>
void fill_gpu(T *v, T value, int n) {
    auto block_dim = 192ul;
    auto grid_dim = n/block_dim + (n%block_dim ? 1 : 0);

    fill<T><<<grid_dim, block_dim>>>(v, value, n);
}

void write_to_file(int nx, int ny, double* data, int mpi_size, int mpi_rank) {
    // collect the global solution to the root rank
    auto block_size = nx*(ny-2); // discard first and last rows
    std::vector<double> data_global(mpi_size*block_size);
    MPI_Gather(data+nx,          block_size, MPI_DOUBLE,
               &data_global[0], block_size, MPI_DOUBLE,
               0, MPI_COMM_WORLD);

    if(mpi_rank==0) {
        FILE* output = fopen("output.bin", "w");
        fwrite(&data_global[0], sizeof(double), mpi_size* nx * (ny-2), output);
        fclose(output);

        std::ofstream fid("output.bov");
        fid << "TIME: 0.0" << std::endl;
        fid << "DATA_FILE: output.bin" << std::endl;
        fid << "DATA_SIZE: " << nx << ", " << mpi_size*(ny-2) << ", 1" << std::endl;;
        fid << "DATA_FORMAT: DOUBLE" << std::endl;
        fid << "VARIABLE: phi" << std::endl;
        fid << "DATA_ENDIAN: LITTLE" << std::endl;
        fid << "CENTERING: nodal" << std::endl;
        fid << "BRICK_SIZE: 1.0 1.0 1.0" << std::endl;
    }
}
